
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloKernel() {
  printf("hello from block %d, thread %d\n",
         blockIdx.x, threadIdx.x);
}


int main() {
  // launch kernel 2 blocks 3 threads each
  helloKernel<<<2, 4>>>();
  hipDeviceSynchronize(); // wait for kernel to finish

  // host print 
  std::cout <<"hello from the CPU !\n";
  return 0; 
}