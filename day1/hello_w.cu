//"C:/Program Files/NVIDIA GPU Computing Toolkit/CUDA/v11.7/bin/nvcc.exe",







#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c)
{
  int index = threadIdx.x;
  c[index] = a[index] + b[index];
}

int main()
{
  const int size = 5;
  int h_a[size] = {1, 2, 3, 4, 5};
  int h_b[size] = {10, 20, 30, 40, 50};
  int h_c[size];

  int *d_a, *d_b, *d_c;
  hipMalloc((void **)&d_a, size * sizeof(int));
  hipMalloc((void **)&d_b, size * sizeof(int));
  hipMalloc((void **)&d_c, size * sizeof(int));

  hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

  add<<<1, size>>>(d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "Result: ";
  for (int i = 0; i < size; i++)
  {
    std::cout << h_c[i] << " ";
  }
  std::cout << std::endl;

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
